#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include "timer.h"

#define BLOCK_SIZE 16
#define GRID_SIZE 256

#define CUDA_CALL(x) {if((x) != hipSuccess){ \
  printf("CUDA error at %s:%d\n",__FILE__,__LINE__); \
  printf("  %s\n", hipGetErrorString(hipGetLastError())); \
  exit(EXIT_FAILURE);}}


__constant__ int dev_nCentroids;
__constant__ int dev_size;


int PALETTE_BYTES = 0;
int IMAGE_BYTES = 0;  

__constant__ int dev_RedCentroid[20];
__constant__ int dev_GreenCentroid[20];
__constant__ int dev_BlueCentroid[20];

//RGB Values
bool loadPalette(char* filename, int nCentroids, int* redCentroid, int* greenCentroid, int*  blueCentroid) {

	FILE *imageFile;
	int length = 0;

	imageFile = fopen(filename,"r");
	if (imageFile == NULL) {
		return false;
	} else {
		for (int i = 0; i < nCentroids; i++) {

			
			redCentroid[i] = fgetc(imageFile);
			greenCentroid[i] = fgetc(imageFile);
			blueCentroid[i] = fgetc(imageFile);
			printf("%d, %d, %d\n",redCentroid[i], greenCentroid[i], blueCentroid[i] );
			length++;
		}
		fclose(imageFile);
		printf("\n");
		printf("Tamaño de la paleta: %d\n", length);
		return true;
	}
}


bool loadRawImage(char* filename, int* r, int* g, int* b, int size) {
	FILE *imageFile;
	imageFile = fopen(filename, "r");

	if (imageFile == NULL) {
		return false;
	} else {
		for (int i = 0; i < size; i++) {

			r[i] = fgetc(imageFile);
			g[i] = fgetc(imageFile);
			b[i] = fgetc(imageFile);
		}
		fclose(imageFile);

		/*for(int j = 0; j < h * w; j++) {
			printf("%d, %d, %d ", r[j], g[j], b[j]);
		}*/
		return true;
	}
}

bool writeRawImage(char* filename, int* labelArray, int* redCentroid, int* greenCentroid, int* blueCentroid, int size){
	FILE *imageFile;
	imageFile = fopen(filename, "wb");

	if(imageFile == NULL) {
		return false;
	} else {
		for (int i = 0; i < size; i++) {
			fputc((char) redCentroid[labelArray[i]], imageFile);
			fputc((char) greenCentroid[labelArray[i]], imageFile);
			fputc((char) blueCentroid[labelArray[i]], imageFile);
		}
		fclose(imageFile);
		return true;
	}
}

__global__ void clearPaletteArrays(int *dev_sumRed,int *dev_sumGreen,int *dev_sumBlue, int* dev_pixelClusterCounter, int* dev_tempRedCentroid, int* dev_tempGreenCentroid, int* dev_tempBlueCentroid ) {

	// 1 block, 16x16 threads
	int threadID = threadIdx.x + threadIdx.y * blockDim.x;

	if(threadID < dev_nCentroids) {

		// nCentroids long
		dev_sumRed[threadID] = 0;
		dev_sumGreen[threadID] = 0;
		dev_sumBlue[threadID] = 0;
		dev_pixelClusterCounter[threadID] = 0;
		dev_tempRedCentroid[threadID] = 0;
		dev_tempGreenCentroid[threadID] = 0;
		dev_tempBlueCentroid[threadID] = 0;
	}
}


__global__ void clearLabelArray(int *dev_labelArray){

	// Global thread index
	int threadID = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;
	if(threadID < dev_size) {
		dev_labelArray[threadID] = 0;
	}
}

__global__ void getClusterLabel(int *dev_Red,int *dev_Green,int *dev_Blue,int *dev_labelArray) {


	// Global thread index
	int threadID = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;

	float min = 500.0, value;
	int index = 0;


	if(threadID < dev_size) {
		for(int i = 0; i < dev_nCentroids; i++) {
			value = sqrtf(powf((dev_Red[threadID]-dev_RedCentroid[i]),2.0) + powf((dev_Green[threadID]-dev_GreenCentroid[i]),2.0) + powf((dev_Blue[threadID]-dev_BlueCentroid[i]),2.0));

			if(value < min){
				
				min = value;
				
				index = i;
			}
		}
		dev_labelArray[threadID] = index;

	}
}

__global__ void sumCluster(int *dev_Red,int *dev_Green,int *dev_Blue,int *dev_sumRed,int *dev_sumGreen,int *dev_sumBlue,int *dev_labelArray,int *dev_pixelClusterCounter) {

	// Global thread index
	int threadID = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;


	if(threadID < dev_size) {
		int currentLabelArray = dev_labelArray[threadID];
		int currentRed = dev_Red[threadID];
		int currentGreen = dev_Green[threadID];
		int currentBlue = dev_Blue[threadID];
		atomicAdd(&dev_sumRed[currentLabelArray], currentRed);
		atomicAdd(&dev_sumGreen[currentLabelArray], currentGreen);
		atomicAdd(&dev_sumBlue[currentLabelArray], currentBlue);
		atomicAdd(&dev_pixelClusterCounter[currentLabelArray], 1);
	}
}

__global__ void newCentroids(int *dev_tempRedCentroid, int *dev_tempGreenCentroid, int *dev_tempBlueCentroid,int* dev_sumRed, int *dev_sumGreen,int *dev_sumBlue, int* dev_pixelClusterCounter) {

	// 1 block , 16*16 threads
	int threadID = threadIdx.x + threadIdx.y * blockDim.x;

	if(threadID < dev_nCentroids) {
		int currentPixelCounter = dev_pixelClusterCounter[threadID];
		int sumRed = dev_sumRed[threadID];
		int sumGreen = dev_sumGreen[threadID];
		int sumBlue = dev_sumBlue[threadID];

		//new RGB Centroids' values written in global memory
		dev_tempRedCentroid[threadID] = (int)(sumRed/currentPixelCounter);
		dev_tempGreenCentroid[threadID] = (int)(sumGreen/currentPixelCounter);
		dev_tempBlueCentroid[threadID] = (int)(sumBlue/currentPixelCounter);
	}

}

int main(int argc, char *argv[]) {

		// init device
		hipSetDevice(0);
		hipDeviceSynchronize();
		hipDeviceSynchronize();

		
		char *inputFile, *outputFile, *palette;
		int *red, *green, *blue, *redCentroid, *greenCentroid, *blueCentroid;
		int *dev_Red, *dev_Green, *dev_Blue, *dev_tempRedCentroid, *dev_tempGreenCentroid, *dev_tempBlueCentroid;
		int *labelArray, *dev_labelArray;
		int width, height, nCentroids, nIterations,size;
		int *pixelClusterCounter, *dev_pixelClusterCounter;
		int *sumRed, *sumGreen, *sumBlue;
		int *dev_sumRed, *dev_sumGreen, *dev_sumBlue;

		
		if (argc > 7) {
			inputFile = argv[1];
			outputFile = argv[2];
			width = atoi(argv[3]);
			height = atoi(argv[4]);
			palette = argv[5];
			nCentroids = atoi(argv[6]);  
			if(nCentroids > 256)
				nCentroids = 256;
			nIterations = atoi(argv[7]);
			if(nIterations > 15)
				nIterations = 15;

		} else {
			printf("  Compilar: kmeans.cu <inputfile.raw> <outputfile.raw> nRows nCols paleta nCentroides nItarationes \n");
			return 0;
		}

		
		IMAGE_BYTES = width * height * sizeof(int);
		PALETTE_BYTES = nCentroids * sizeof(int);
		size = width * height;


		printf("Image: %s\n",inputFile);
		printf("Width: %d, Height: %d\n", width, height);
		printf("#Clusters: %d, #Iterations: %d\n", nCentroids, nIterations);

		red = static_cast<int *>(malloc(IMAGE_BYTES));
		green = static_cast<int *>(malloc(IMAGE_BYTES));
		blue = static_cast<int *>(malloc(IMAGE_BYTES));
		redCentroid = static_cast<int *>(malloc(PALETTE_BYTES));
		greenCentroid = static_cast<int *>(malloc(PALETTE_BYTES));
		blueCentroid = static_cast<int *>(malloc(PALETTE_BYTES));
		labelArray = static_cast<int *>(malloc(IMAGE_BYTES));
		sumRed = static_cast<int*>(malloc(PALETTE_BYTES));
		sumGreen = static_cast<int*>(malloc(PALETTE_BYTES));
		sumBlue = static_cast<int*>(malloc(PALETTE_BYTES));
		pixelClusterCounter = static_cast<int*>(malloc(PALETTE_BYTES));

		printf("Initial Centroids: \n");
		if(loadPalette(palette, nCentroids, redCentroid, greenCentroid, blueCentroid)) {
		} else {
			printf("Unable to set Initial Centroids.\n");
		}

		printf("Image loading...\n");
		if (loadRawImage(inputFile, red, green, blue, size)) {
			printf("Image loaded!\n");
		} else {
			printf("NOT loaded!\n");
			return -1;
		}

		printf("\n");

		

		if(IMAGE_BYTES == 0 || PALETTE_BYTES == 0) {
			return -1;
		}

		CUDA_CALL(hipMalloc((void**) &dev_Red, IMAGE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_Green, IMAGE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_Blue, IMAGE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_tempRedCentroid, PALETTE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_tempGreenCentroid, PALETTE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_tempBlueCentroid, PALETTE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_labelArray, IMAGE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_sumRed, PALETTE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_sumGreen, PALETTE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_sumBlue, PALETTE_BYTES));
		CUDA_CALL(hipMalloc((void**) &dev_pixelClusterCounter, PALETTE_BYTES));

		// copy host CPU memory to GPU
		CUDA_CALL(hipMemcpy(dev_Red, red, IMAGE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_Green, green, IMAGE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_Blue, blue, IMAGE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_tempRedCentroid, redCentroid,PALETTE_BYTES,hipMemcpyHostToDevice ));
		CUDA_CALL(hipMemcpy(dev_tempGreenCentroid, greenCentroid,PALETTE_BYTES,hipMemcpyHostToDevice ));
		CUDA_CALL(hipMemcpy(dev_tempBlueCentroid, blueCentroid,PALETTE_BYTES,hipMemcpyHostToDevice ));
		CUDA_CALL(hipMemcpy(dev_labelArray, labelArray, IMAGE_BYTES, hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(dev_sumRed, sumRed, PALETTE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_sumGreen, sumGreen, PALETTE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_sumBlue, sumBlue, PALETTE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dev_pixelClusterCounter, pixelClusterCounter, PALETTE_BYTES, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_RedCentroid), redCentroid, PALETTE_BYTES));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_GreenCentroid), greenCentroid, PALETTE_BYTES));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_BlueCentroid), blueCentroid, PALETTE_BYTES));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_nCentroids),&nCentroids, sizeof(int)));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_size), &size, sizeof(int)));


		// Clearing centroids on host
		for(int i = 0; i < nCentroids; i++) {
			redCentroid[i] = 0;
			greenCentroid[i] = 0;
			blueCentroid[i] = 0;
		}

		// Defining grid size

		int BLOCK_X, BLOCK_Y;
		BLOCK_X = ceil(width/BLOCK_SIZE);
		BLOCK_Y = ceil(height/BLOCK_SIZE);
		if(BLOCK_X > GRID_SIZE)
			BLOCK_X = GRID_SIZE;
		if(BLOCK_Y > GRID_SIZE)
			BLOCK_Y = GRID_SIZE;

		//2D Grid
		//Minimum number of threads that can handle width¡height pixels
	 	dim3 dimGRID(BLOCK_X,BLOCK_Y);
	 	//2D Block
	 	//Each dimension is fixed
		dim3 dimBLOCK(BLOCK_SIZE,BLOCK_SIZE);

		//Starting timer
		GpuTimer timer;
		timer.Start();

		printf("Launching K-Means Kernels..	\n");
		//Iteration of kmeans algorithm
		for(int i = 0; i < nIterations; i++) {


			clearPaletteArrays<<<1, dimBLOCK>>>(dev_sumRed, dev_sumGreen, dev_sumBlue, dev_pixelClusterCounter, dev_tempRedCentroid, dev_tempGreenCentroid, dev_tempBlueCentroid);

			clearLabelArray<<<dimGRID, dimBLOCK>>>(dev_labelArray);
			
			getClusterLabel<<< dimGRID, dimBLOCK >>> (dev_Red, dev_Green, dev_Blue,dev_labelArray);


			sumCluster<<<dimGRID, dimBLOCK>>> (dev_Red, dev_Green, dev_Blue, dev_sumRed, dev_sumGreen, dev_sumBlue, dev_labelArray,dev_pixelClusterCounter);


			newCentroids<<<1,dimBLOCK >>>(dev_tempRedCentroid, dev_tempGreenCentroid, dev_tempBlueCentroid, dev_sumRed, dev_sumGreen, dev_sumBlue, dev_pixelClusterCounter);


			CUDA_CALL(hipMemcpy(redCentroid, dev_tempRedCentroid, PALETTE_BYTES,hipMemcpyDeviceToHost));
			CUDA_CALL(hipMemcpy(greenCentroid, dev_tempGreenCentroid, PALETTE_BYTES,hipMemcpyDeviceToHost));
			CUDA_CALL(hipMemcpy(blueCentroid, dev_tempBlueCentroid, PALETTE_BYTES,hipMemcpyDeviceToHost));

			CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_RedCentroid), redCentroid, PALETTE_BYTES));
			CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_GreenCentroid), greenCentroid, PALETTE_BYTES));
			CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_BlueCentroid), blueCentroid, PALETTE_BYTES));
			timer.Stop();
		}


		CUDA_CALL(hipMemcpy(labelArray, dev_labelArray, IMAGE_BYTES, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(sumRed, dev_sumRed, PALETTE_BYTES, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(sumGreen, dev_sumGreen, PALETTE_BYTES, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(sumBlue, dev_sumBlue, PALETTE_BYTES, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(pixelClusterCounter, dev_pixelClusterCounter, PALETTE_BYTES, hipMemcpyDeviceToHost));

		printf("Kmeans tiempo: %f msecs.\n", timer.Elapsed());
		printf("\n");

		
		  int counter = 0;

		printf("Label Array:\n");
		for(int i = 0; i < (size); i++) {
			//printf("%d\n", labelArray[i]);
			counter++;
		}
		printf("printing counter %d\n", counter);
		counter = 0;

		printf("Pallete:\n");
		for(int j = 0; j < nCentroids; j++) {
			printf("r: %u g: %u b: %u \n", sumRed[j], sumGreen[j], sumBlue[j]);
			counter++;
		}

		printf("\n");

		printf("Pixels por centroides:\n");
		for(int k = 0; k < nCentroids; k++){
			printf("%d centroid: %d pixels\n", k, pixelClusterCounter[k]);
		}

		printf("\n");



		printf("Nuevos centroides:\n");
		for(int i = 0; i < nCentroids; i++) {

			printf("%d, %d, %d \n", redCentroid[i], greenCentroid[i], blueCentroid[i]);
		}


		
		printf("Image ...\n");

		if (writeRawImage(outputFile,labelArray, redCentroid, greenCentroid,  blueCentroid,  size)) {
			printf("Image procesada \n");
		} else {
			printf("No procesada\n");
			return -1;
		}

		free(red);
		free(green);
		free(blue);
		free(redCentroid);
		free(greenCentroid);
		free(blueCentroid);
		free(labelArray);
		free(sumRed);
		free(sumGreen);
		free(sumBlue);
		free(pixelClusterCounter);

		CUDA_CALL(hipFree(dev_Red));
		CUDA_CALL(hipFree(dev_Green));
		CUDA_CALL(hipFree(dev_Blue));
		CUDA_CALL(hipFree(dev_tempRedCentroid));
		CUDA_CALL(hipFree(dev_tempGreenCentroid));
		CUDA_CALL(hipFree(dev_tempBlueCentroid));
		CUDA_CALL(hipFree(dev_labelArray));
		CUDA_CALL(hipFree(dev_sumRed));
		CUDA_CALL(hipFree(dev_sumGreen));
		CUDA_CALL(hipFree(dev_sumBlue));
		CUDA_CALL(hipFree(dev_pixelClusterCounter));

		printf("Elapsed time...\n");
		return 0;
}